/*
 * ECE 5720 Parallel Computing Final Project
 * KMP parallel on MPI
 * Feng Qi, fq26
 * Ying Zong, yz887
 * Cornell University
 *
 * Compile : /usr/local/cuda-8.0/bin/nvcc -arch=compute_35 -o cuda kmp-cuda.cu
 * Run     : ./cuda
 */


#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <time.h>


// build the kmp table for the subsequent operations
void preKMP(char* pattern, int func[]) {
    int m = strlen(pattern);
    int k;
    func[0] = -1;
    for (int i = 1; i < m; i++) {
        k = func[i - 1];
        while (k >= 0) {
            if (pattern[k] == pattern[i - 1]) {
                 break;
            }
            else {
                k = func[k];
            }
        }
        func[i] = k + 1;
    }
}

// Kernel function. Implement the KMP algorithm
__global__ void KMP(char* pattern, char* target, int func[], int answer[], int pattern_length, int target_length) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int i = pattern_length * index;
    int j = pattern_length * (index + 2) - 1;
    
    if(i > target_length) {
        return;
    }

    if(j > target_length) {
        j = target_length;
    }

    int k = 0;        
    while (i < j) {
        if (k == -1) {
            i++;
            k = 0;
        } else if (target[i] == pattern[k]) {
            i++;
            k++;
            if (k == pattern_length) {
                answer[i - pattern_length] = i - pattern_length;
                i = i - k + 1;
            }
        }
        else {
            k = func[k];
        }
    }
    return;
}

int main(int argc, char* argv[]) {
    int M = 4;
    int n = 100000;
    int m = 3;    
    char* target = (char*)malloc(n * sizeof(char));
    char* pattern = (char*)malloc(m * sizeof(char));
    printf("----- This is parallel results using KMP Algo on CUDA. -----\n");
    FILE * file = fopen( "data.txt" , "r");
    int CurrentIndex = 0;
    while (CurrentIndex < 2) {
        if (CurrentIndex == 0) {
            fgets(target, n+1, file);
        } else if (CurrentIndex == 1) {
            fgets(pattern, m+1, file);
        }
        CurrentIndex++;
    }
    int target_length = strlen(target);
    int pattern_length = strlen(pattern);

    fclose(file);

    char *d_target;
    char *d_pattern;

    int *func;
    int *answer;

    func = new int[target_length];
    answer = new int[target_length];

    int *d_func;
    int *d_answer;
    for(int i = 0; i < target_length; i++) {
        answer[i] = -1;
    }     

    preKMP(pattern, func);

    hipEvent_t start, stop;
    float elapsedTime;

    hipEventCreate( &start ); 
    hipEventCreate( &stop );

    hipEventRecord( start, 0 );

    hipMalloc((void **)&d_target, target_length * sizeof(char));
    hipMalloc((void **)&d_pattern, pattern_length * sizeof(char));
    hipMalloc((void **)&d_func, target_length * sizeof(int));
    hipMalloc((void **)&d_answer, target_length * sizeof(int));

    hipMemcpy(d_target, target, target_length * sizeof(char), hipMemcpyHostToDevice);
    hipMemcpy(d_pattern, pattern, pattern_length * sizeof(char), hipMemcpyHostToDevice);
    hipMemcpy(d_func, func, target_length * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_answer, answer, target_length * sizeof(int), hipMemcpyHostToDevice);

    KMP<<<(target_length / pattern_length + M)/M, M>>>(d_pattern, d_target ,d_func, d_answer, pattern_length, target_length);

    hipMemcpy(answer, d_answer, target_length * sizeof(int), hipMemcpyDeviceToHost);

    hipEventRecord( stop, 0 );
    hipEventSynchronize( stop );
    hipEventElapsedTime( &elapsedTime, start, stop );
    hipEventDestroy(start); 
    hipEventDestroy(stop);

    printf("When the target length is %d, pattern length is %d, the elapsed time is %0.3f ms.\n", target_length, pattern_length, elapsedTime); 

    for(int i = 0; i < target_length; i++) {
        if (answer[i] != -1) {
            printf("Find a matching substring starting at: %d.\n", i);
        }
    }

    hipFree(d_target); 
    hipFree(d_pattern); 
    hipFree(d_func); 
    hipFree(d_answer);
    free(target);
    free(pattern);
    return 0;
}