/*
 * ECE 5720 Parallel Computing Final Project
 * KMP parallel on MPI
 * Feng Qi, fq26
 * Ying Zong, yz887
 * Cornell University
 *
 * Compile : /usr/local/cuda-8.0/bin/nvcc -arch=compute_35 -o cuda kmp-cuda.cu
 * Run     : ./cuda
 */


#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <time.h>

// build the kmp table for the subsequent operations
void preKMP(char* pattern, int func[]) {
    int m = strlen(pattern);
    int k;
    func[0] = -1;
    for (int i = 1; i < m; i++) {
        k = func[i - 1];
        while (k >= 0) {
            if (pattern[k] == pattern[i - 1]) {
                 break;
            }
            else {
                k = func[k];
            }
        }
        func[i] = k + 1;
    }
}

// Kernel function. Implement the KMP algorithm
__global__ void KMP(char* pattern, char* target, int func[], int answer[], int pattern_length, int target_length) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int i = pattern_length * index;
    int j = pattern_length * (index + 2) - 1;
    
    if(i > target_length) {
        return;
    }

    if(j > target_length) {
        j = target_length;
    }

    int k = 0;        
    while (i < j) {
        if (k == -1) {
            i++;
            k = 0;
        } else if (target[i] == pattern[k]) {
            i++;
            k++;
            if (k == pattern_length) {
                answer[i - pattern_length] = i - pattern_length;
                i = i - k + 1;
            }
        }
        else {
            k = func[k];
        }
    }
    return;
}

int main(int argc, char* argv[]) {
    const int target_length = 260;
    const int pattern_length = 4;
    int M = 4;

    char *target;
    char *pattern;
    target = (char*)malloc(target_length * sizeof(char));
    pattern = (char*)malloc(pattern_length * sizeof(char));

    char* dict = "abcdefghijklmnopqrstuvwxyz";
    for (int i = 0; i < target_length; i++) {
        target[i] = dict[i%26];
    }
    for (int j = 0; j < pattern_length; j++) {
        pattern[j] = dict[j%26];
    }
    char *d_target;
    char *d_pattern;
    printf("----- This is parallel results using KMP Algo on CUDA. -----\n");
    // printf("The target length is: %d, the pattern length is %d.\n", target_length, pattern_length);
    // printf("The target string is: \n");
    // printf("%s\n", target);
    // printf("The pattern string is: \n");
    // printf("%s\n", pattern);
    int *func;
    int *answer;

    func = new int[target_length];
    answer = new int[target_length];

    int *d_func;
    int *d_answer;
    for(int i = 0; i < target_length; i++) {
        answer[i] = -1;
    }     

    preKMP(pattern, func);

    hipEvent_t start, stop;
    float elapsedTime;

    hipEventCreate( &start ); 
    hipEventCreate( &stop );

    hipEventRecord( start, 0 );

    hipMalloc((void **)&d_target, target_length * sizeof(char));
    hipMalloc((void **)&d_pattern, pattern_length * sizeof(char));
    hipMalloc((void **)&d_func, target_length * sizeof(int));
    hipMalloc((void **)&d_answer, target_length * sizeof(int));

    hipMemcpy(d_target, target, target_length * sizeof(char), hipMemcpyHostToDevice);
    hipMemcpy(d_pattern, pattern, pattern_length * sizeof(char), hipMemcpyHostToDevice);
    hipMemcpy(d_func, func, target_length * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_answer, answer, target_length * sizeof(int), hipMemcpyHostToDevice);

    KMP<<<(target_length / pattern_length + M)/M, M>>>(d_pattern, d_target ,d_func, d_answer, pattern_length, target_length);

    hipMemcpy(answer, d_answer, target_length * sizeof(int), hipMemcpyDeviceToHost);

    hipEventRecord( stop, 0 );
    hipEventSynchronize( stop );
    hipEventElapsedTime( &elapsedTime, start, stop );
    hipEventDestroy(start); 
    hipEventDestroy(stop);

    printf("When the target length is %d, pattern length is %d, the elapsed time is %0.3f ms.\n", target_length, pattern_length, elapsedTime); 

    for(int i = 0; i < target_length; i++) {
        if (answer[i] != -1) {
            printf("Find a matching substring starting at: %d.\n", i);
        }
    }

    hipFree(d_target); 
    hipFree(d_pattern); 
    hipFree(d_func); 
    hipFree(d_answer);

    return 0;
}
